#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <math.h>
#include <cmath>
#include <stdio.h>
#include <string.h>

#include "Gaussian.h"

const int TILE_SIZE = 32;

bool checkForError(hipError_t error, char* funcName);

// Scale kernel takes in each row and divides all other elements in row with the index element. For eg: in the first row index element is (1,1)...2nd row it is (2,2)...so on..
// It divides all other elements in the row with their corresponding index elements
__global__ void ScaleRowKernel(float* in, float* out, unsigned int numRows, unsigned int numCols, int pivot){

	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;

	if (row < numRows && col < numCols){

		if (row == pivot && col != pivot) out[row*numCols + col] = in[row*numCols + col] / in[pivot*numCols + pivot];  // The index (pivot) element for scaling ..I'm reading it from the input matrix !!

		if (row == pivot && col == pivot){ out[pivot*numCols + pivot] = 1; }  // Instead of dividing the index element by itself in the row, I simply make it 1 because it has to be....
		//else out[row*numCols + col] = in[row*numCols + col];
	}
}


// Subtraction kernel takes in each row....  multiplies the multiplying factor with the elements of pivot row and subtracts the elements of other rows with the pivot row
// Similar implementation to CPU
__global__ void SubtractionKernel(float* in, float* out, unsigned int numRows, unsigned int numCols, int pivot){

	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;

	if (row < numRows && col < numCols){
		if (row != pivot){
			out[row*numCols + col] = in[row*numCols + col] - (in[row*numCols + pivot] * in[pivot*numCols + col]);
		}
		else out[row*numCols + col] = in[pivot*numCols + col];
	}
}

bool GaussianEliminationGPU(float** matrix, unsigned int numberOfRows, unsigned int numberOfColumns, float** outputMatrix, bool partialPivot){


	hipError_t status;
	bool error;
	int bytes = numberOfRows * numberOfColumns * sizeof(float);
	float scaleFactor = 0;
	float* out;
	float* in;

	// Initializing the values
	hipMalloc((void**)&out, bytes);
	hipMalloc((void**)&in, bytes);

	status = hipGetLastError();
	error = checkForError(status, "At hipMalloc");
	if (!error){
		hipFree(out);
		hipFree(in);
		return false;
	}

	//transfer of data from host to device
	for (int r = 0; r<numberOfRows; r++){
		hipMemcpy(&out[r*numberOfColumns], matrix[r], numberOfColumns*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&in[r*numberOfColumns], matrix[r], numberOfColumns*sizeof(float), hipMemcpyHostToDevice);
	}
	status = hipGetLastError();
	error = checkForError(status, "At hipMemcpy");
	if (!error){
		hipFree(out);
		hipFree(in);
		return false;
	}

	// Defining Grid and block
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);   // 2D block
	status = hipGetLastError();
	error = checkForError(status, "At dimBlock");
	if (!error){
		hipFree(out);
		hipFree(in);
		return false;
	}

	//Using 2 Dimensional grid
	int gridSize1 = (int)ceil((((float)numberOfColumns) / (TILE_SIZE)));
	int gridSize2 = (int)ceil((((float)numberOfRows) / (TILE_SIZE)));
	dim3 dimGrid(gridSize1, gridSize2);
	status = hipGetLastError();
	error = checkForError(status, "At dimGrid");
	if (!error){
		hipFree(out);
		hipFree(in);
		return false;
	}

	//Scaling each row and performing elementary row wise operations
	for (int p = 0; p<numberOfRows; p++){ // p is the pivot row

		ScaleRowKernel << < dimGrid, dimBlock >> >(in, out, numberOfRows, numberOfColumns, p);
		hipDeviceSynchronize();
		hipDeviceSynchronize();
		status = hipGetLastError();
		error = checkForError(status, "At ScaleRowKernel");
		if (!error){
			hipFree(out);
			hipFree(in);
			return false;
		}
		// After each operation on a row, I'm copying the output matrix (out) into input matrix (in)..so that next scaling row will have updated input matrix
		hipMemcpy(in, out, numberOfRows*numberOfColumns*sizeof(float), hipMemcpyDeviceToDevice);


		SubtractionKernel << < dimGrid, dimBlock >> >(in, out, numberOfRows, numberOfColumns, p);
		hipDeviceSynchronize();
		hipDeviceSynchronize();
		status = hipGetLastError();
		error = checkForError(status, "At SubtractionKernel");
		if (!error){
			printf("Error at Subtraction kernel row number : %d", p);
			hipFree(out);
			hipFree(in);
			return false;
		}
		// After each operation on a row, I'm copying the output matrix (out) into input matrix (in)..so that next scaling row will have updated input matrix
		hipMemcpy(in, out, numberOfRows*numberOfColumns*sizeof(float), hipMemcpyDeviceToDevice);

		hipDeviceSynchronize();
		hipDeviceSynchronize();
	}

	for (int r = 0; r<numberOfRows; r++){
		hipMemcpy(outputMatrix[r], &out[r*numberOfColumns],
			numberOfColumns*sizeof(float), hipMemcpyDeviceToHost);
	}
	
	status = hipGetLastError();
	error = checkForError(status, "After cudaMemcopy to host");
	if (!error){
		hipFree(out);
		hipFree(in);
		return false;
	}


	hipFree(out);
	hipFree(in);
	return true;
}

bool checkForError(hipError_t error, char* funcName){
	bool success;
	if (error != hipSuccess){
		success = false;
		printf("Device error at %s \n", funcName);
		printf("%s", hipGetErrorString);
	}
	else{
		success = true;
	}
	return success;
}
